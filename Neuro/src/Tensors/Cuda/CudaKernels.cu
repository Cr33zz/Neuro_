#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "Tensors/Cuda/CudaKernels.h"

__global__ void leakyRelu(int inputLen, const float* __restrict input, float alpha, float* __restrict result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputLen)
        result[i] = input[i] > 0 ? input[i] : (alpha * input[i]);
}

__global__ void leakyReluGrad(int inputLen, const float* __restrict output, const float* __restrict outputGradient, float alpha, float* __restrict result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputLen)
        result[i] = (output[i] > 0 ? 1 : alpha) * outputGradient[i];
}

__global__ void div(int inputLen, const float* __restrict input, float v, float* __restrict result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputLen)
        result[i] = input[i] / v;
}

__device__ int getIndex(int w, int h, int d, int n, int dim0, int dim0dim1, int dim0dim1dim2)
{
    return w + h * dim0 + d * dim0dim1 + n * dim0dim1dim2;
}

__device__ void getDims(int width, int height, int depth, int& dim0, int& dim0dim1, int& dim0dim1dim2)
{
    dim0 = width;
    dim0dim1 = width * height;
    dim0dim1dim2 = width * height * depth;
}

__global__ void addBroadcast(float alpha, const float* __restrict t1, int t1Width, int t1Height, int t1Depth, int t1Batch, float beta, const float* __restrict t2, int t2Width, int t2Height, int t2Depth, int t2Batch, float* __restrict output, int outputWidth, int outputHeight, int outputDepth, int outputBatch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    int outputLen = outputWidth * outputHeight * outputDepth * outputBatch;

    if (i >= outputLen)
        return;
    
    int outputDim0, outputDim0Dim1, outputDim0Dim1Dim2;
    getDims(outputWidth, outputHeight, outputDepth, outputDim0, outputDim0Dim1, outputDim0Dim1Dim2);

    int t1Dim0, t1Dim0Dim1, t1Dim0Dim1Dim2;
    getDims(t1Width, t1Height, t1Depth, t1Dim0, t1Dim0Dim1, t1Dim0Dim1Dim2);

    int t2Dim0, t2Dim0Dim1, t2Dim0Dim1Dim2;
    getDims(t2Width, t2Height, t2Depth, t2Dim0, t2Dim0Dim1, t2Dim0Dim1Dim2);

    int w = i % outputWidth;
    int h = (i / outputDim0) % outputHeight;
    int d = (i / outputDim0Dim1) % outputDepth;
    int n = i / outputDim0Dim1Dim2;

    int t1N = n % t1Batch;
    int t2N = n % t2Batch;
    int t1D = d % t1Depth;
    int t2D = d % t2Depth;
    int t1H = h % t1Height;
    int t2H = h % t2Height;
    int t1W = w % t1Width;
    int t2W = w % t2Width;

    output[i] = alpha * t1[getIndex(t1W, t1H, t1D, t1N, t1Dim0, t1Dim0Dim1, t1Dim0Dim1Dim2)] + beta * t2[getIndex(t2W, t2H, t2D, t2N, t2Dim0, t2Dim0Dim1, t2Dim0Dim1Dim2)];
}

__global__ void adamStep(int inputLen, float* __restrict parameterDev, float* __restrict gradientDev, float* __restrict mGradDev, float* __restrict vGradDev, float batchSize, float lr, float beta1, float beta2, float epsilon)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputLen)
    {
        gradientDev[i] /= batchSize;
        mGradDev[i] = beta1 * mGradDev[i] + (1 - beta1) * gradientDev[i];
        vGradDev[i] = beta2 * vGradDev[i] + (1 - beta2) * gradientDev[i] * gradientDev[i];
        parameterDev[i] -= mGradDev[i] / (sqrt(vGradDev[i]) + epsilon) * lr;
        gradientDev[i] = 0;
    }
}

__global__ void sgdStep(int inputLen, float* __restrict parameterDev, float* __restrict gradientDev, float batchSize, float lr)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputLen)
    {
        parameterDev[i] -= gradientDev[i] / batchSize * lr;
        gradientDev[i] = 0;
    }
}

template<class F>
__global__ void map(int inputLen, const float* __restrict input, F f, float* __restrict result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputLen)
        result[i] = f(input[i]);
}

namespace Neuro
{
    void CudaKernels::LeakyReLU(const dim3& blocks, const dim3& threads, int inputLen, const float* inputDev, float alpha, float* outputDev)
    {
        leakyRelu<<<blocks, threads>>>(inputLen, inputDev, alpha, outputDev);
        hipDeviceSynchronize();
    }

    void CudaKernels::LeakyReLUGradient(const dim3& blocks, const dim3& threads, int inputLen, const float* outputDev, const float* outputGradientDev, float alpha, float* inputGradientDev)
    {
        leakyReluGrad<<<blocks, threads>>>(inputLen, outputDev, outputGradientDev, alpha, inputGradientDev);
        hipDeviceSynchronize();
    }

    void CudaKernels::Div(const dim3& blocks, const dim3& threads, int inputLen, const float* inputDev, float v, float* outputDev)
    {
        div<<<blocks, threads>>>(inputLen, inputDev, v, outputDev);
        hipDeviceSynchronize();
    }

    void CudaKernels::AddBroadcast(const dim3& blocks, const dim3& threads, float alpha, const float* t1Dev, int t1Width, int t1Height, int t1Depth, int t1Batch, float beta, const float* t2Dev, int t2Width, int t2Height, int t2Depth, int t2Batch, float* outputDev, int outputWidth, int outputHeight, int outputDepth, int outputBatch)
    {
        addBroadcast<<<blocks, threads>>>(alpha, t1Dev, t1Width, t1Height, t1Depth, t1Batch, beta, t2Dev, t2Width, t2Height, t2Depth, t2Batch, outputDev, outputWidth, outputHeight, outputDepth, outputBatch);
        hipDeviceSynchronize();
    }

    void CudaKernels::AdamStep(const dim3& blocks, const dim3& threads, int inputLen, float* parameterDev, float* gradientDev, float* mGradDev, float* vGradDev, float batchSize, float lr, float beta1, float beta2, float epsilon)
    {
        adamStep<<<blocks, threads>>>(inputLen, parameterDev, gradientDev, mGradDev, vGradDev, batchSize, lr, beta1, beta2, epsilon);
        hipDeviceSynchronize();
    }

    void CudaKernels::SgdStep(const dim3& blocks, const dim3& threads, int inputLen, float* parameterDev, float* gradientDev, float batchSize, float lr)
    {
        sgdStep<<<blocks, threads>>>(inputLen, parameterDev, gradientDev, batchSize, lr);
        hipDeviceSynchronize();
    }
}